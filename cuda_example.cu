#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void addKernel(int* c, const int* a, const int* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int* c, const int* a, const int* b, int size) {
    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;

    // 為三個向量（兩個輸入，一個輸出）分配 GPU 緩衝區
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    // 將輸入向量從主機內存複製到 GPU 緩衝區。
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // 在 GPU 上啟動一個內核，每個元素都有一個線程。
    // 2 是計算塊的數量， (size + 1) / 2 是塊中的線程數

    addKernel<<<2, (size + 1) / 2>>>(dev_c, dev_a, dev_b, size);
    
    // cudaDeviceSynchronize 等待內核完成，然後返回
    // any errors encountered during the launch.
    hipDeviceSynchronize();

    // 將輸出向量從 GPU 緩衝區復製到主機內存。
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

int main(int argc, char** argv) {
    const int arraySize = 5;
    const int a[arraySize] = {  1,  2,  3,  4,  5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    addWithCuda(c, a, b, arraySize);

    printf("{1, 2, 3, 4, 5} + {10, 20, 30, 40, 50} = {%d, %d, %d, %d, %d}\n", c[0], c[1], c[2], c[3], c[4]);

    hipDeviceReset();

    return 0;
}
